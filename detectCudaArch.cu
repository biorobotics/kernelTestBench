#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int main()
{
    hipDeviceProp_t dP;
    float minComputeCapability = 3.0;

    int rc = hipGetDeviceProperties(&dP, 0);
    if(rc != hipSuccess) {
        hipError_t error = hipGetLastError();
        printf("CUDA error: %s", hipGetErrorString(error));
        return rc;
    }
    if((dP.major+(dP.minor/10)) < minComputeCapability) {
        printf("Min Compute Capability of %2.1f required:  %d.%d found\n Not Building CUDA Code", minComputeCapability, dP.major, dP.minor);
        return 1; /* Failure */
    } else {
        printf("%d%d", dP.major, dP.minor);
        return 0; /* Success */
    }
}
