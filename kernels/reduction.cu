#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "reduction.cuh"


__device__ void warp_reduce_max( volatile float smem[64])
{

	smem[threadIdx.x] = smem[threadIdx.x+32] > smem[threadIdx.x] ? 
						smem[threadIdx.x+32] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+16] > smem[threadIdx.x] ? 
						smem[threadIdx.x+16] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+8] > smem[threadIdx.x] ? 
						smem[threadIdx.x+8 ] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+4] > smem[threadIdx.x] ? 
						smem[threadIdx.x+4] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+2] > smem[threadIdx.x] ? 
						smem[threadIdx.x+2] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+1] > smem[threadIdx.x] ? 
						smem[threadIdx.x+1] : smem[threadIdx.x]; DEBUG_SYNC;

}

__device__ void warp_reduce_min(volatile float smem[64])
{

	smem[threadIdx.x] = smem[threadIdx.x+32] < smem[threadIdx.x] ? 
						smem[threadIdx.x+32] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+16] < smem[threadIdx.x] ? 
						smem[threadIdx.x+16] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+8] < smem[threadIdx.x] ? 
						smem[threadIdx.x+8] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+4] < smem[threadIdx.x] ? 
						smem[threadIdx.x+4] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+2] < smem[threadIdx.x] ? 
						smem[threadIdx.x+2] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+1] < smem[threadIdx.x] ? 
						smem[threadIdx.x+1] : smem[threadIdx.x]; DEBUG_SYNC;

}

template<int threads>
__global__ void find_min_max_dynamic(float* in, float* out, int n, int start_adr, int num_blocks)
{

	__shared__ float smem_min[64];
	__shared__ float smem_max[64];

	int tid = threadIdx.x + start_adr;

	float max = -inf;
	float min = inf;
	float val;


	// tail part
	int mult = 0;
	for(int i = 1; mult + tid < n; i++)
	{
		val = in[tid + mult];
	
		min = val < min ? val : min;
		max = val > max ? val : max;

		mult = int_mult(i,threads);
	}

	// previously reduced MIN part
	mult = 0;
	int i;
	for(i = 1; mult+threadIdx.x < num_blocks; i++)
	{
		val = out[threadIdx.x + mult];

		min = val < min ? val : min;
		
		mult = int_mult(i,threads);
	}

	// MAX part
	for(; mult+threadIdx.x < num_blocks*2; i++)
	{
		val = out[threadIdx.x + mult];

		max = val > max ? val : max;
		
		mult = int_mult(i,threads);
	}


	if(threads == 32)
	{
		smem_min[threadIdx.x+32] = 0.0f;
		smem_max[threadIdx.x+32] = 0.0f;

	}
	
	smem_min[threadIdx.x] = min;
	smem_max[threadIdx.x] = max;

	__syncthreads();

	if(threadIdx.x < 32)
	{
		warp_reduce_min(smem_min);
		warp_reduce_max(smem_max);
	}
	if(threadIdx.x == 0)
	{
		out[blockIdx.x] = smem_min[threadIdx.x]; // out[0] == ans
		out[blockIdx.x + gridDim.x] = smem_max[threadIdx.x]; 
	}


}

template<int els_per_block, int threads>
__global__ void find_min_max(float* in, float* out)
{
	__shared__ float smem_min[64];
	__shared__ float smem_max[64];

	int tid = threadIdx.x + blockIdx.x*els_per_block;

	float max = -inf;
	float min = inf;
	float val;

	const int iters = els_per_block/threads;
	
#pragma unroll
		for(int i = 0; i < iters; i++)
		{

			val = in[tid + i*threads];

			min = val < min ? val : min;
			max = val > max ? val : max;

		}
	
	
	if(threads == 32)
	{
		smem_min[threadIdx.x+32] = 0.0f;
		smem_max[threadIdx.x+32] = 0.0f;
	
	}
	
	smem_min[threadIdx.x] = min;
	smem_max[threadIdx.x] = max;


	__syncthreads();

	if(threadIdx.x < 32)
	{
		warp_reduce_min(smem_min);
		warp_reduce_max(smem_max);
	}
	if(threadIdx.x == 0)
	{
		out[blockIdx.x] = smem_min[threadIdx.x]; // out[0] == ans
		out[blockIdx.x + gridDim.x] = smem_max[threadIdx.x]; 
	}

}
